#include "Symmetria.hpp"
#include "dCSR.cuh"
#include "SemiRingInterface.h"
#include "../source/device/Multiply.cuh"

#include "test_utils.cuh"

//#define DEBUG_TEST

using namespace symmetria;

template <typename T>
dCSR<T> cpu_transpose(dCSR<T>& A)
{
    std::vector<T> h_vals(A.nnz);
    std::vector<unsigned int> h_colinds(A.nnz);
    std::vector<unsigned int> h_rowptrs(A.rows + 1);

    CUDA_CHECK(hipMemcpy(h_vals.data(), A.data, sizeof(T)*h_vals.size(),
                            hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_colinds.data(), A.col_ids, sizeof(unsigned int )*A.nnz,
                            hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_rowptrs.data(), A.row_offsets, sizeof(unsigned int) * (A.rows + 1),
                            hipMemcpyDeviceToHost));

	unsigned int num_cols = A.cols;
	unsigned int num_rows = A.rows;

    std::vector<T> h_trans_vals(A.nnz);
    std::vector<unsigned int> h_trans_colinds(A.nnz);
    std::vector<unsigned int> h_trans_rowptrs(A.cols + 1);

	// Initialize row pointers for the transposed matrix
    h_trans_rowptrs.resize(num_cols + 1, 0);

    // Step 1: Compute the number of elements in each column (which becomes row in transposed matrix)
    for (int i = 0; i < h_colinds.size(); ++i) {
        h_trans_rowptrs[h_colinds[i] + 1]++;
    }

    // Step 2: Compute the cumulative row pointer for the transposed matrix
    for (int i = 0; i < num_cols; ++i) {
        h_trans_rowptrs[i + 1] += h_trans_rowptrs[i];
    }

    // Step 3: Initialize the values and columns for the transposed matrix
    h_trans_vals.resize(h_vals.size());
    h_trans_colinds.resize(h_colinds.size());

    // Step 4: Fill the values and columns for the transposed matrix
    std::vector<int> col_counts(num_cols, 0);  // Temporary array to track element position in rows

    for (int row = 0; row < num_rows; ++row) {
        for (int j = h_rowptrs[row]; j < h_rowptrs[row + 1]; ++j) {
            int col = h_colinds[j];
            int dest_pos = h_trans_rowptrs[col] + col_counts[col];

            h_trans_vals[dest_pos] = h_vals[j];
            h_trans_colinds[dest_pos] = row;
            
            col_counts[col]++;
        }
    }

    dCSR<T> A_t;
    A_t.alloc(num_cols, num_rows, A.nnz);

    CUDA_CHECK(hipMemcpy(A_t.data, h_trans_vals.data(), sizeof(T)*A.nnz, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(A_t.col_ids, h_trans_colinds.data(), sizeof(unsigned int)*A.nnz, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(A_t.row_offsets, h_trans_rowptrs.data(), sizeof(unsigned int)*(A.cols + 1), hipMemcpyHostToDevice));

    return A_t;
}
        



int main(int argc, char ** argv)
{
    typedef unsigned int IT;
    typedef double DT;

    symmetria_init();

    {

        const uint32_t m = 16;
        const uint32_t n = 16;
        const uint32_t nnz = 16;

        std::shared_ptr<ProcMap> proc_map = std::make_shared<ProcMap>(n_pes, MPI_COMM_WORLD);

        DistSpMat1DBlockRow<IT, DT> A(m, n, nnz, proc_map);
        symmetria::io::read_mm<IT, DT>("../test/test_matrices/n16.mtx", A);

        MPI_Barrier(MPI_COMM_WORLD);

        MPI_Barrier(MPI_COMM_WORLD);

        dCSR<DT> A_dcsr = make_dCSR_from_distspmat_outofplace<DT>(A);

        dCSR<DT> A_dcsr_correct = cpu_transpose(A_dcsr);

        auto A_t = transpose_outofplace(A_dcsr);

#ifdef DEBUG_TEST
        logptr->OFS()<<"A"<<std::endl;
        dump_dCSR_to_log(logptr, A_dcsr);
        logptr->OFS()<<"At_correct"<<std::endl;
        dump_dCSR_to_log(logptr, A_dcsr_correct);
        logptr->OFS()<<"At_actual"<<std::endl;
        dump_dCSR_to_log(logptr, A_t);
#endif

        bool is_correct = (A_t == A_dcsr_correct);

        TEST_CHECK(is_correct);

        dealloc(A_t);
        dealloc(A_dcsr_correct);
    }

    TEST_SUCCESS("Transpose");
    symmetria_finalize();
	
    return 0;
}
