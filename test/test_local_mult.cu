#include "hip/hip_runtime.h"
#include "Symmetria.hpp"
#include "dCSR.cuh"
#include "CSR.cuh"
#include "CPU_SpGEMM.h"
#include "SemiRingInterface.h"
#include "../source/device/Multiply.cuh"

#include "TestDriver.hpp"

//#define DEBUG_TEST
#define EPS 0.001

using namespace symmetria;
using namespace symmetria::testing;


class TestLocalMult : public TestDriver<TestLocalMult>
{
public:

    bool run_test_impl(TestParams& test)
    {
        typedef unsigned int IT;
        typedef double DT;

        const uint32_t m = test.rows;
        const uint32_t n = test.cols;
        const uint32_t nnz = test.nnz;
        const std::string name(test.name);
        const std::string path(std::string("../test/test_matrices/")+test.name+".mtx");

        std::shared_ptr<ProcMap> proc_map = std::make_shared<ProcMap>(n_pes, MPI_COMM_WORLD);

        /* Read in matrix */
        DistSpMat1DBlockRow<IT, DT> A(m, n, nnz, proc_map);
        symmetria::io::read_mm<IT, DT>(path.c_str(), A);
        MPI_Barrier(MPI_COMM_WORLD);
        DEBUG_PRINT("Done reading");

        /* Fetch local submatrices */
        dCSR<DT> A_dcsr = make_dCSR_from_distspmat_outofplace<DT>(A);
        dCSR<DT> A_t = transpose_outofplace(A_dcsr);


#ifdef DEBUG_TEST
        dump_dCSR_to_log(logptr, A_dcsr);
        dump_dCSR_to_log(logptr, A_t);
#endif

        /* GPU multiply */
        using Semiring = PlusTimesSemiring<DT>;
        IT nnz_C;
        auto d_C = local_spgemm_galatic<Semiring>(A_dcsr, A_t, nnz_C);

        /* Do CPU multiply for correctness check */
        CSR<DT> h_A;
        CSR<DT> h_A_t;
        CSR<DT> h_C_csr;

        convert(h_A, A_dcsr);
        convert(h_A_t, A_t);
        CUDA_CHECK(hipDeviceSynchronize());

        Semiring semiring;
        Mult_CPU<Semiring>(h_A, h_A_t, h_C_csr, semiring);

        CooTriples<IT, DT> h_C_triples(h_C_csr.data.get(), h_C_csr.col_ids.get(), 
                                        h_C_csr.row_offsets.get(), 
                                        h_C_csr.nnz, h_C_csr.rows);
#ifdef DEBUG_TEST
        h_C_triples.dump_to_log(logptr, "Correct output");
#endif

        TEST_CHECK(((size_t)nnz_C == h_C_csr.nnz));
        TEST_CHECK(compare(h_C_triples, d_C, nnz_C));

        CUDA_CHECK(hipFree(d_C));

        return true;
    }


    template <typename IT, typename DT>
    bool compare(CooTriples<IT, DT>& h_correct_triples, 
                 std::tuple<IT, IT, DT> * d_triples, const IT nnz)
    {

        std::vector<std::tuple<IT, IT, DT>> h_actual_triples(nnz);

        CUDA_CHECK(hipMemcpy(h_actual_triples.data(), d_triples, sizeof(std::tuple<IT, IT, DT>)* nnz,
                                hipMemcpyDeviceToHost));

        auto triple_comp = [](auto& t1, auto& t2) 
        {
            return (std::get<0>(t1)==std::get<0>(t2) &&
                    std::get<1>(t1)==std::get<1>(t2) &&
                    fabs(std::get<2>(t1) - std::get<2>(t2)) < EPS);
        };

        return testing::compare_vectors(h_correct_triples, h_actual_triples, triple_comp);
    }


};






int main(int argc, char ** argv)
{
    int test_id = -1;
    if (argc > 1)
        test_id = std::atoi(argv[1]);

    symmetria_init();
    {
        TestDriver<TestLocalMult> manager("../test/test_configs.json", "Local Multiply", test_id);
        manager.run_tests();
    }
    symmetria_finalize();
	
    return 0;
}
